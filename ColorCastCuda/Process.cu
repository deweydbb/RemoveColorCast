#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include ""
#include "cudart_platform.h"


extern "C" {
    #include "Image.h"
    #include "Tiff.h"
}

// code adapted from: https://stackoverflow.com/questions/5731863/mapping-a-numeric-range-onto-another
// maps a given value in one range into another range
__device__ double mapDouble(double input, double input_start, double input_end, double output_start, double output_end) {
    return output_start + ((output_end - output_start) / (input_end - input_start)) * (input - input_start);
}

// super fancy custom function because: double abs(double) has multiple definitions
// so this is my HIGH tech work around
__device__ double absoluteVal(double val) {
    if (val < 0) {
        return val * -1;
    }

    return val;
}

// returns the dampened r,g, or b value of a color based on the avg value of
// the color and a grayness rating from 0 - 1. With 1 being true gray and 0
// being the opposite of grey
__device__ int dampenColor(int col, double avg, double grayness, double power) {
    double diff = absoluteVal(col - avg);
    // the amount to move towards the avg value of the color
    // (how much to move towards a true gray)
    double change = diff * pow(grayness, power);

    // because diff is absolute valued it is necessary to check
    // if the original color is greater or less than the average
    // to determine if it is necessary to add or subtract change
    // from the original value
    if (col > avg) {
        return col - rint(change);
    }

    return col + (int) rint(change);
}

// thread responsible for processing one pixel of the image
// bytesPerChannel specifies whether file store rgb values in 8 or 16 bit integers.
// max is the pointer 1 after the end of the pixel data. Becaue each thread block has a fixed number of threads
// there is one block that will have excces threads. It is necessary to make sure these threads do nothing. 
__global__ void processPixel(unsigned char* data, unsigned int offset, double power, int bytesPerChannel, int isLittle, unsigned int max) {
    unsigned int pixelNum = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int startPtr = offset + (pixelNum * 3 * bytesPerChannel);
    // check to make sure startPtr is a valid pointer to pixel data
    if (startPtr < max) {
        // values hardcoded because having local variables is faster than calling malloc on the gpu
        // to create an array
        int red = 0;
        int green = 0;
        int blue = 0;
        
        if (bytesPerChannel == 1) {
            red = data[startPtr];
            green = data[startPtr + 1];
            blue = data[startPtr + 2];
        } else if (isLittle) {
            red = data[startPtr];
            red += data[startPtr + 1] << 8;
            green = data[startPtr + 2];
            green += data[startPtr + 3] << 8;
            blue = data[startPtr + 4];
            blue += data[startPtr + 5] << 8;
        } else {
            red = data[startPtr] << 8;
            red += data[startPtr + 1];
            green = data[startPtr + 2] << 8;
            green += data[startPtr + 3];
            blue = data[startPtr + 4] << 8;
            blue += data[startPtr + 5];
        }
        
        double grayness = abs(red - green) + abs(red - blue) + abs(blue - green);

        int maxRange = 65536 * 2;
        if (bytesPerChannel == 1) {
            maxRange = 255 * 2;
        }
        
        // maps grayness from range of [0, maxRange] to [0, 1]
        grayness = mapDouble(grayness, 0, maxRange, 0, 1);
        // reverses range. Now 1 is true gray and 0 is opposite of true gray
        grayness = 1 - grayness;
        
        // calculates the average rgb value of the color
        double avg = (double) (red + green + blue) / 3;
        // returns the nomalized color by "dampening" the rgb values individually
        red = dampenColor(red, avg, grayness, power);
        green = dampenColor(green, avg, grayness, power);
        blue = dampenColor(blue, avg, grayness, power);
        // set processed rgb values back out to gpu global memory
        if (bytesPerChannel == 1) {
            data[startPtr] = red;
            data[startPtr + bytesPerChannel] = green;
            data[startPtr + (bytesPerChannel * 2)] = blue;
        }
        else if (isLittle) {
            // little endian 16 bit
            data[startPtr] = red;
            data[startPtr + 1] = red >> 8;
            data[startPtr + 2] = green;
            data[startPtr + 3] = green >> 8;
            data[startPtr + 4] = blue;
            data[startPtr + 5] = blue >> 8;
        }
        else {
            // big endian 16 bit
            data[startPtr] = red >> 8;
            data[startPtr + 1] = red;
            data[startPtr + 2] = green >> 8;
            data[startPtr + 3] = green;
            data[startPtr + 4] = blue >> 8;
            data[startPtr + 5] = blue;
        }
        
    }
}

// processes any image on the gpu that is not a tiff
// copies over pixel data to gpu and creates a thread for every pixel
int handleImage(char* imagePath, char* outputPath, double power) {
    // load in the image
    Image* img = getImage(imagePath);
    const int NUM_CHANNELS = 3;
    if (img == NULL) {
        return -1;
    }

    unsigned char* d_pix;
    unsigned int numPix = img->width * img->height;
    // allocate memory on the gpu
    hipError_t err = hipMalloc(&d_pix, numPix * NUM_CHANNELS * sizeof(char));
    if (err != hipSuccess) {
        printf("Error on malloc %s\n", hipGetErrorString(err));
        return -1;
    }
    // copy over pixel data to gpu 
    err = hipMemcpy(d_pix, img->pix, numPix * NUM_CHANNELS * sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error on memcopy htd %s\n", hipGetErrorString(err));
        return -1;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (numPix + threadsPerBlock - 1) / threadsPerBlock;
    // create threads on gpu to process each individual pixel
    processPixel <<<blocksPerGrid, threadsPerBlock>>> (d_pix, 0, power, 1, 1, numPix * 3);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error on process pixels %s\n", hipGetErrorString(err));
        return -1;
    }
    // copy over pixel data from gpu to cpu
    err = hipMemcpy(img->pix, d_pix, numPix * NUM_CHANNELS * sizeof(char), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error on memcopy dth %s\n", hipGetErrorString(err));
        return -1;
    }
    // free memory on the gpu
    err = hipFree(d_pix);
    if (err != hipSuccess) {
        printf("Error on free in main %s\n", hipGetErrorString(err));
        return -1;
    }
    // write image to output file
    writeImage(img, outputPath);
    // return 0 indicating success
    return 0;
}

// handles a single strip tiff. Copies over just the pixel data to the gpu
// creates gpu thread for each pixel
int handleSingleStrip(Tiff* tiff, double power, char* outputPath) {
    unsigned long numPixels = getWidth(tiff) * getHeight(tiff);
    unsigned long pixelStartOffset = tiff->stripOffsets[0];
    unsigned int numBytes = tiff->bytesPerStrip[0];
    unsigned char* d_pix;
    // allocate space on gpu for pixel data of tiff
    hipError_t err = hipMalloc(&d_pix, tiff->bytesPerStrip[0] * sizeof(char));
    if (err != hipSuccess) {
        printf("Error on malloc %s\n", hipGetErrorString(err));
        return -1;
    }
    // copy over pixel data to gpu
    err = hipMemcpy(d_pix, tiff->data + pixelStartOffset, numBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error on memcopy htd %s\n", hipGetErrorString(err));
        return -1;
    }

    int threadsPerBlock = 256;
    // creates enough blockes so there is one thread per pixel
    int blocksPerGrid = (numPixels + threadsPerBlock - 1) / threadsPerBlock;
    int bytesPerChannel = tiff->bitsPerSample / 8;
    int isLittle = tiff->isLittle;
    // create threads on gpu
    processPixel <<<blocksPerGrid, threadsPerBlock>>> (d_pix, 0, power, bytesPerChannel, isLittle, numBytes);
    // check for error on threads in gpu
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error on process pixels %s\n", hipGetErrorString(err));
        return -1;
    }
    // copy processed pixel data from gpu to cpu
    err = hipMemcpy(tiff->data + pixelStartOffset, d_pix, numBytes * sizeof(char), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error on memcopy dth %s\n", hipGetErrorString(err));
        return -1;
    }
    // free memory on gpu
    err = hipFree(d_pix);
    if (err != hipSuccess) {
        printf("Error on free in main %s\n", hipGetErrorString(err));
        return -1;
    }
    // write tiff to output file
    writeTiff(tiff, outputPath);
    // return 0 indicating success
    return 0;
}

// the handeling of multi stripped tiffs is handled separately because there is no guarantee that the strips will be 
// placed continuously throughout the file so it faster to copy the entire file all at once to the gpu than copy over
// each strip. This not does not make sense for singely stripped tiffs, where the pixels are guaranteed to be stored 
// continuously in the file. creates thread for each pixel.
int handleMultiStrips(Tiff* tiff, double power, char* outputPath) {
    unsigned char* d_pix;

    // malloc enough gpu memory for the entire tiff file
    hipError_t err = hipMalloc(&d_pix, tiff->dataLen);
    if (err != hipSuccess) {
        printf("Error on malloc %s\n", hipGetErrorString(err));
        return -1;
    }
    // copy over entire tiff file to gpu
    err = hipMemcpy(d_pix, tiff->data, tiff->dataLen, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error on memcopy htd %s\n", hipGetErrorString(err));
        return -1;
    }
    
    int isLittle = tiff->isLittle;
    int bytesPerChannel = tiff->bitsPerSample / 8;
    int threadsPerBlock = 256;
    // loop through each strip of the tiff 
    for (int i = 0; i < tiff->numStrips; i++) {
        int numPixelsInStrip = tiff->bytesPerStrip[i] / (3 * bytesPerChannel);
        int blocksPerGrid = (numPixelsInStrip + threadsPerBlock - 1) / threadsPerBlock;
        // max pointer value of the strip
        unsigned int max = tiff->stripOffsets[i] + tiff->bytesPerStrip[i];
        // processPixel is an async call so the next strip can be setup relatively quickly
        processPixel << <blocksPerGrid, threadsPerBlock >> > (d_pix, tiff->stripOffsets[i], power, bytesPerChannel, isLittle, max);
        // check for error while processing pixels
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Error on process pixels %s\n", hipGetErrorString(err));
            return -1;
        }
    }
    // copy tiff file from gpu to cpu
    err = hipMemcpy(tiff->data, d_pix, tiff->dataLen, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error on memcopy dth %s\n", hipGetErrorString(err));
        return -1;
    }
    // free gpu memory
    err = hipFree(d_pix);
    if (err != hipSuccess) {
        printf("Error on free in main %s\n", hipGetErrorString(err));
        return -1;
    }
    // write the tiff to the output file
    writeTiff(tiff, outputPath);
    // return 0 indicating success
    return 0;
}

