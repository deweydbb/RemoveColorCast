#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "Process.cuh"
#include ""
#include "cudart_platform.h"


extern "C" {
    #include "Image.h"
    #include "Tiff.h"
}

// code adapted from: https://stackoverflow.com/questions/5731863/mapping-a-numeric-range-onto-another
// maps a given value in one range into another range
__device__ double mapDouble(double input, double input_start, double input_end, double output_start, double output_end) {
    return output_start + ((output_end - output_start) / (input_end - input_start)) * (input - input_start);
}

__device__ double absoluteVal(double val) {
    if (val < 0) {
        return val * -1;
    }

    return val;
}

// returns the dampened r,g, or b value of a color based on the avg value of
// the color and a grayness rating from 0 - 1. With 1 being true gray and 0
// being the opposite of grey
__device__ int dampenColor(int col, double avg, double grayness, double power) {
    double diff = absoluteVal(col - avg);
    // the amount to move towards the avg value of the color
    // (how much to move towards a true gray)
    double change = diff * pow(grayness, power);

    // because diff is absolute valued it is necessary to check
    // if the original color is greater or less than the average
    // to determine if it is necessary to add or subtract change
    // from the original value
    if (col > avg) {
        return col - rint(change);
    }

    return col + (int) rint(change);
}

__global__ void processPixel(unsigned char* data, unsigned int offset, double power, int bytesPerChannel, int isLittle, unsigned int max) {
    unsigned int pixelNum = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int startPtr = offset + (pixelNum * 3 * bytesPerChannel);

    if (startPtr < max) {
        int red = data[startPtr];
        int green = data[startPtr + 1];
        int blue = data[startPtr + 2];
        if (bytesPerChannel == 2 && isLittle) {
            red = data[startPtr];
            red += data[startPtr + 1] << 8;
            green = data[startPtr + 2];
            green += data[startPtr + 3] << 8;
            blue = data[startPtr + 4];
            blue += data[startPtr + 5] << 8;
        } 
        else if (bytesPerChannel == 2) {
            red = data[startPtr] << 8;
            red += data[startPtr + 1];
            green = data[startPtr + 2] << 8;
            green += data[startPtr + 3];
            blue = data[startPtr + 4] << 8;
            blue += data[startPtr + 5];
        }
        
        double grayness = abs(red - green) + abs(red - blue) + abs(blue - green);

        int maxRange = 65536 * 2;
        if (bytesPerChannel == 1) {
            maxRange = 255 * 2;
        }

        // maps grayness from range of [0, maxRange] to [0, 1]
        grayness = mapDouble(grayness, 0, maxRange, 0, 1);
        // reverses range. Now 1 is true gray and 0 is opposite of true gray
        grayness = 1 - grayness;
        
        // calculates the average rgb value of the color
        double avg = (double) (red + green + blue) / 3;
        // returns the nomalized color by "dampening" the rgb values individually
        red = dampenColor(red, avg, grayness, power);
        green = dampenColor(green, avg, grayness, power);
        blue = dampenColor(blue, avg, grayness, power);
       
        if (bytesPerChannel == 1) {
            data[startPtr] = red;
            data[startPtr + bytesPerChannel] = green;
            data[startPtr + (bytesPerChannel * 2)] = blue;
        }
        else if (isLittle) {
            data[startPtr] = red;
            data[startPtr + 1] = red >> 8;
            data[startPtr + 2] = green;
            data[startPtr + 3] = green >> 8;
            data[startPtr + 4] = blue;
            data[startPtr + 5] = blue >> 8;
        }
        else {
            data[startPtr] = red >> 8;
            data[startPtr + 1] = red;
            data[startPtr + 2] = green >> 8;
            data[startPtr + 3] = green;
            data[startPtr + 4] = blue >> 8;
            data[startPtr + 5] = blue;
        }
        
    }
}

int handleImage(char* imagePath, char* outputPath, double power) {
    Image* img = getImage(imagePath);
    const int NUM_CHANNELS = 3;
    if (img == NULL) {
        return -1;
    }

    unsigned char* d_pix;
    unsigned int numPix = img->width * img->height;

    hipError_t err = hipMalloc(&d_pix, numPix * NUM_CHANNELS * sizeof(char));
    if (err != hipSuccess) {
        printf("Error on malloc %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(d_pix, img->pix, numPix * NUM_CHANNELS * sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error on memcopy htd %s\n", hipGetErrorString(err));
        return -1;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (numPix + threadsPerBlock - 1) / threadsPerBlock;

    processPixel <<<blocksPerGrid, threadsPerBlock>>> (d_pix, 0, power, 1, 1, numPix * 3);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error on process pixels %s\n", hipGetErrorString(err));
        return -1;
    }


    err = hipMemcpy(img->pix, d_pix, numPix * NUM_CHANNELS * sizeof(char), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error on memcopy dth %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipFree(d_pix);
    if (err != hipSuccess) {
        printf("Error on free in main %s\n", hipGetErrorString(err));
        return -1;
    }

    writeImage(img, outputPath);

    return 0;
}

int handleSingleStrip(Tiff* tiff, double power, char* outputPath) {
    unsigned long numPixels = getWidth(tiff) * getHeight(tiff);
    unsigned long pixelStartOffset = tiff->stripOffsets[0];
    unsigned int numBytes = tiff->bytesPerStrip[0];
    unsigned char* d_pix;

    clock_t start = clock();

    hipError_t err = hipMalloc(&d_pix, tiff->bytesPerStrip[0] * sizeof(char));
    if (err != hipSuccess) {
        printf("Error on malloc %s\n", hipGetErrorString(err));
        return -1;
    }
    
    err = hipMemcpy(d_pix, tiff->data + pixelStartOffset, numBytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error on memcopy htd %s\n", hipGetErrorString(err));
        return -1;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (numPixels + threadsPerBlock - 1) / threadsPerBlock;
    int bytesPerChannel = tiff->bitsPerSample / 8;
    int isLittle = tiff->isLittle;

    processPixel <<<blocksPerGrid, threadsPerBlock>>> (d_pix, 0, power, bytesPerChannel, isLittle, numBytes);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error on process pixels %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(tiff->data + pixelStartOffset, d_pix, numBytes * sizeof(char), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error on memcopy dth %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipFree(d_pix);
    if (err != hipSuccess) {
        printf("Error on free in main %s\n", hipGetErrorString(err));
        return -1;
    }

    //printf("total process data time: %.f\n", (double)(clock() - start));

    start = clock();
    writeTiff(tiff, outputPath);
    //printf("write time: %.3f\n", (double)(clock() - start) / 1000);
    
    return 0;

}

int handleMultiStrips(Tiff* tiff, double power, char* outputPath) {
    unsigned char* d_pix;

    clock_t start = clock();

    hipError_t err = hipMalloc(&d_pix, tiff->dataLen);
    if (err != hipSuccess) {
        printf("Error on malloc %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMemcpy(d_pix, tiff->data, tiff->dataLen, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("Error on memcopy htd %s\n", hipGetErrorString(err));
        return -1;
    }

    int isLittle = tiff->isLittle;
    int bytesPerChannel = tiff->bitsPerSample / 8;
    int threadsPerBlock = 256;

    for (int i = 0; i < tiff->numStrips; i++) {
        int numPixelsInStrip = tiff->bytesPerStrip[i] / (3 * bytesPerChannel);
        int blocksPerGrid = (numPixelsInStrip + threadsPerBlock - 1) / threadsPerBlock;
        unsigned int max = tiff->stripOffsets[i] + tiff->bytesPerStrip[i];
        processPixel << <blocksPerGrid, threadsPerBlock >> > (d_pix, tiff->stripOffsets[i], power, bytesPerChannel, isLittle, max);

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("Error on process pixels %s\n", hipGetErrorString(err));
            return -1;
        }
    }

    err = hipMemcpy(tiff->data, d_pix, tiff->dataLen, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("Error on memcopy dth %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipFree(d_pix);
    if (err != hipSuccess) {
        printf("Error on free in main %s\n", hipGetErrorString(err));
        return -1;
    }

    //printf("total process data time: %.f\n", (double)(clock() - start));

    start = clock();
    writeTiff(tiff, outputPath);
    //printf("write time: %.3f\n", (double)(clock() - start) / 1000);

    return 0;
}

